#include "hip/hip_runtime.h"
#include "NumericalIntegrator3d.cuh"

#include "common/cuda_memory.cuh"

__constant__ Point3 c_GaussPointsCoordinates[CONSTANTS::MAX_GAUSS_POINTS];
__constant__ double c_GaussPointsWeights[CONSTANTS::MAX_GAUSS_POINTS];
__constant__ int    c_GaussPointsNumber;

__global__ void kSplitCell(int n, Point3 *refinedVertices, int3 *refinedCells, double *refinedCellMeasures, int *originalCells, int2 *refinedVerticesCellsNum,
        const Point3 *tempVertices, const int3 *tempCells, const double *tempCellMeasures, const int *tempOriginalCells)
{
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if(idx < n){
        //data of the triangle to be split
        const int3 triangle = tempCells[idx];
        const Point3 triA = tempVertices[triangle.x];
        const Point3 triB = tempVertices[triangle.y];
        const Point3 triC = tempVertices[triangle.z];
        const double measure = tempCellMeasures[idx];
        const int originalCellIndex = tempOriginalCells[idx];

        //add 3 new vertices on the triangle egdes
        const int vertexIndex = atomicAdd((int*)refinedVerticesCellsNum, 3);
        refinedVertices[vertexIndex] = 0.5 * (triB + triC);
        refinedVertices[vertexIndex + 1] = 0.5 * (triC + triA);
        refinedVertices[vertexIndex + 2] = 0.5 * (triA + triB);

        //create 4 new triangles
        const int cellIndex = atomicAdd((int*)refinedVerticesCellsNum + 1, 4);
        refinedCells[cellIndex] = { vertexIndex + 2, triangle.y, vertexIndex };
        refinedCells[cellIndex + 1] = { vertexIndex, triangle.z, vertexIndex + 1 };
        refinedCells[cellIndex + 2] = { vertexIndex + 1, triangle.x, vertexIndex + 2 };
        refinedCells[cellIndex + 3] = { vertexIndex, vertexIndex + 1, vertexIndex + 2 };

        //determine measure of each new cell
        const double newMeasure = 0.25 * measure;
        refinedCellMeasures[cellIndex] = newMeasure;
        refinedCellMeasures[cellIndex + 1] = newMeasure;
        refinedCellMeasures[cellIndex + 2] = newMeasure;
        refinedCellMeasures[cellIndex + 3] = newMeasure;

        //update correspondence to the original cell index
        originalCells[cellIndex] = originalCellIndex;
        originalCells[cellIndex + 1] = originalCellIndex;
        originalCells[cellIndex + 2] = originalCellIndex;
        originalCells[cellIndex + 3] = originalCellIndex;
    }
}

__global__ void kCountOrCreateTasks(int tasksNum, int refinedCellsNum, int *counter, const int3 *tasks, const int *originalCells, int3 *refinedTasks = nullptr)
{
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

    __shared__ int originalCellsSh[gpuThreadsMax];

	//do not exit the kernel immediately when idx >= tasksNum
	//because the thread might be required to perform loading into shared memory
	for(int blockStart = 0; blockStart < refinedCellsNum; blockStart += gpuThreadsMax){
		if(blockStart + threadIdx.x < refinedCellsNum)
			originalCellsSh[threadIdx.x] = originalCells[blockStart + threadIdx.x];
		__syncthreads();

		if(idx < tasksNum){
			const int3 oldTask = tasks[idx];

			for(int cell = 0; cell < gpuThreadsMax; ++cell)
				if(blockStart + cell < refinedCellsNum){
					const int originalCell = originalCellsSh[cell];

					if(oldTask.x == originalCell){
						int pos = atomicAdd(counter, 1);
						if(refinedTasks)
							refinedTasks[pos] = { blockStart + cell, oldTask.y, (int)idx };
					}
				}
		}

		__syncthreads();
	}
}

__global__ void kSumIntegrationResults(int n, double4 *results, const double4 *refinedResults, const int3 *refinedTasks)
{
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if(idx < n){
        const int3 task = refinedTasks[idx];
        const double4 refinedResult = refinedResults[idx];
        double *originalResult = (double*)(results + task.z);

        atomicAdd(originalResult + 0, refinedResult.x);
        atomicAdd(originalResult + 1, refinedResult.y);
        atomicAdd(originalResult + 2, refinedResult.z);
        atomicAdd(originalResult + 3, refinedResult.w);
    }
}

__global__ void kInitializeOriginalCellIndices(int n, int *originalCells)
{
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if(idx < n)
        originalCells[idx] = idx;
}

NumericalIntegrator3D::NumericalIntegrator3D(const Mesh3D &mesh_, const QuadratureFormula3D &qf_)
    : GaussPointsNum(qf_.weights.size()), mesh(mesh_), qf(qf_)
{
    std::vector<Point3> lCoordinates(GaussPointsNum);

    for(int i = 0; i < GaussPointsNum; ++i){
        lCoordinates[i].x = qf_.coordinates[i].x;
        lCoordinates[i].y = qf_.coordinates[i].y;
        lCoordinates[i].z = 1.0 - qf_.coordinates[i].x - qf_.coordinates[i].y;
    }

    copy_h2const(lCoordinates.data(), c_GaussPointsCoordinates, GaussPointsNum);
    copy_h2const(qf.weights.data(), c_GaussPointsWeights, GaussPointsNum);
    copy_h2const(&GaussPointsNum, &c_GaussPointsNumber, 1);

    errorControlType = error_control_type_enum::automatic_error_control;
}

NumericalIntegrator3D::~NumericalIntegrator3D()
{

}

void NumericalIntegrator3D::setFixedRefinementLevel(int refinementLevel)
{
    errorControlType = error_control_type_enum::fixed_refinement_level;
    meshRefinementLevel = refinementLevel;
}

void NumericalIntegrator3D::prepareTasksAndRefineWholeMesh(const deviceVector<int3> &simpleNeighborsTasks, const deviceVector<int3> &attachedNeighborsTasks, const deviceVector<int3> &notNeighborsTasks)
{
    int2 verticesCellsNum = { mesh.getVertices().size, mesh.getCells().size };

    if(!meshRefinementLevel){
        //copy mesh vertices, cells and measures as is
        refinedVertices.allocate(verticesCellsNum.x);
        copy_d2d(mesh.getVertices().data, refinedVertices.data, verticesCellsNum.x);

        refinedCells.allocate(verticesCellsNum.y);
        copy_d2d(mesh.getCells().data, refinedCells.data, verticesCellsNum.y);
        
        refinedCellMeasures.allocate(verticesCellsNum.y);
        copy_d2d(mesh.getCellMeasures().data, refinedCellMeasures.data, verticesCellsNum.y);
        
        //copy tasks and prepare vectors for results
        if(simpleNeighborsTasks.size){
            refinedSimpleNeighborsTasks.allocate(simpleNeighborsTasks.size);
            copy_d2d(simpleNeighborsTasks.data, refinedSimpleNeighborsTasks.data, simpleNeighborsTasks.size);
            d_simpleNeighborsResults.allocate(simpleNeighborsTasks.size);
        }

        if(attachedNeighborsTasks.size){
            refinedAttachedNeighborsTasks.allocate(attachedNeighborsTasks.size);
            copy_d2d(attachedNeighborsTasks.data, refinedAttachedNeighborsTasks.data, attachedNeighborsTasks.size);
            d_simpleNeighborsResults.allocate(attachedNeighborsTasks.size);
        }

        if(notNeighborsTasks.size){
            refinedNotNeighborsTasks.allocate(notNeighborsTasks.size);
            copy_d2d(notNeighborsTasks.data, refinedNotNeighborsTasks.data, notNeighborsTasks.size);
            d_simpleNeighborsResults.allocate(notNeighborsTasks.size);
        }

        return;
    }

    int refinedVerticesNum = verticesCellsNum.x + verticesCellsNum.y * ((1 << (2 * meshRefinementLevel)) - 1);
    refinedVertices.allocate(refinedVerticesNum);
    tempVertices.allocate(refinedVerticesNum);

    int refinedCellsNum = (1 << (2 * meshRefinementLevel)) * verticesCellsNum.y;
    refinedCells.allocate(refinedCellsNum);
    tempCells.allocate(refinedCellsNum);
    refinedCellMeasures.allocate(refinedCellsNum);
    tempCellMeasures.allocate(refinedCellsNum);

    //vectors for indices of original triangles (with respect to the refined triangles)
    deviceVector<int> originalCells, tempOriginalCells;
    originalCells.allocate(refinedCellsNum);
    tempOriginalCells.allocate(refinedCellsNum);

    //initialize vectors using original data (will then be moved to the temporary buffers)
    copy_d2d(mesh.getVertices().data, refinedVertices.data, verticesCellsNum.x);
    copy_d2d(mesh.getCells().data, refinedCells.data, verticesCellsNum.y);
    copy_d2d(mesh.getCellMeasures().data, refinedCellMeasures.data, verticesCellsNum.y);

    //initialize number of indices of original cells with 0,1,2,...,ncells-1
    unsigned int blocks = blocksForSize(verticesCellsNum.y);
    kInitializeOriginalCellIndices<<<blocks, gpuThreads>>>(verticesCellsNum.y, originalCells.data);

    int2 *refinedVerticesCellsNum;
    allocate_device(&refinedVerticesCellsNum, 1);
    copy_h2d(&verticesCellsNum, refinedVerticesCellsNum, 1);

    for(int i = 0; i < meshRefinementLevel; ++i){
        std::swap(refinedVertices.data, tempVertices.data);
        std::swap(refinedCells.data, tempCells.data);
        std::swap(refinedCellMeasures.data, tempCellMeasures.data);
        std::swap(originalCells.data, tempOriginalCells.data);

        //vertex list from the previous iteration forms the first part of the new vertex list
        //leave the vertex count equal to the size of the previous vertex list and set the cell count to zero
        copy_d2d(tempVertices.data, refinedVertices.data, verticesCellsNum.x);
        zero_value_device((int*)refinedVerticesCellsNum + 1, 1);

        blocks = blocksForSize(verticesCellsNum.y);
        kSplitCell<<<blocks, gpuThreads>>>(verticesCellsNum.y, refinedVertices.data, refinedCells.data, refinedCellMeasures.data, originalCells.data,
                refinedVerticesCellsNum, tempVertices.data, tempCells.data, tempCellMeasures.data, tempOriginalCells.data);

        hipDeviceSynchronize();
        copy_d2h(refinedVerticesCellsNum, &verticesCellsNum, 1);
    }

    //update tasks
    int *taskCount;
    int hostTaskCount;
    allocate_device(&taskCount, 1);

    if(simpleNeighborsTasks.size){
        zero_value_device(taskCount, 1);
        blocks = blocksForSize(simpleNeighborsTasks.size, gpuThreadsMax);

        kCountOrCreateTasks<<<blocks, gpuThreadsMax>>>(simpleNeighborsTasks.size, verticesCellsNum.y, taskCount, simpleNeighborsTasks.data, originalCells.data);

        hipDeviceSynchronize();
        copy_d2h(taskCount, &hostTaskCount, 1);

        refinedSimpleNeighborsTasks.allocate(hostTaskCount);
        d_simpleNeighborsResults.allocate(hostTaskCount);
        zero_value_device(taskCount, 1);
        kCountOrCreateTasks<<<blocks, gpuThreadsMax>>>(simpleNeighborsTasks.size, verticesCellsNum.y, taskCount, simpleNeighborsTasks.data, originalCells.data, refinedSimpleNeighborsTasks.data);
    }

    if(attachedNeighborsTasks.size){
        zero_value_device(taskCount, 1);
        blocks = blocksForSize(attachedNeighborsTasks.size, gpuThreadsMax);
        
        kCountOrCreateTasks<<<blocks, gpuThreadsMax>>>(attachedNeighborsTasks.size, verticesCellsNum.y, taskCount, attachedNeighborsTasks.data, originalCells.data);

        hipDeviceSynchronize();
        copy_d2h(taskCount, &hostTaskCount, 1);

        refinedAttachedNeighborsTasks.allocate(hostTaskCount);
        d_attachedNeighborsResults.allocate(hostTaskCount);
        zero_value_device(taskCount, 1);
        kCountOrCreateTasks<<<blocks, gpuThreadsMax>>>(attachedNeighborsTasks.size, verticesCellsNum.y, taskCount, attachedNeighborsTasks.data, originalCells.data, refinedAttachedNeighborsTasks.data);
    }

    if(notNeighborsTasks.size){
        zero_value_device(taskCount, 1);
        blocks = blocksForSize(notNeighborsTasks.size, gpuThreadsMax);
        
        kCountOrCreateTasks<<<blocks, gpuThreadsMax>>>(notNeighborsTasks.size, verticesCellsNum.y, taskCount, notNeighborsTasks.data, originalCells.data);

        hipDeviceSynchronize();
        copy_d2h(taskCount, &hostTaskCount, 1);

        refinedNotNeighborsTasks.allocate(hostTaskCount);
        d_notNeighborsResults.allocate(hostTaskCount);
        zero_value_device(taskCount, 1);
        kCountOrCreateTasks<<<blocks, gpuThreadsMax>>>(notNeighborsTasks.size, verticesCellsNum.y, taskCount, notNeighborsTasks.data, originalCells.data, refinedNotNeighborsTasks.data);
    }

    hipDeviceSynchronize();
    printf("Refined mesh contains %d vertices and %d cells. Number of tasks: simple neighbors - %d, attached neighbors - %d, non-neighbors - %d\n",
            verticesCellsNum.x, verticesCellsNum.y, refinedSimpleNeighborsTasks.size, refinedAttachedNeighborsTasks.size, refinedNotNeighborsTasks.size);

    free_device(refinedVerticesCellsNum);
    free_device(taskCount);
}

void NumericalIntegrator3D::gatherResults(deviceVector<double4> &results, neighbour_type_enum neighborType) const
{
    int3 *refinedTasks;
    int refinedTasksSize;
    double4 *refinedResults;
    
    switch (neighborType)
    {
    case neighbour_type_enum::simple_neighbors:
        refinedTasks = refinedSimpleNeighborsTasks.data;
        refinedTasksSize = refinedSimpleNeighborsTasks.size;
        refinedResults = d_simpleNeighborsResults.data;
        break;
    case neighbour_type_enum::attached_neighbors:
        refinedTasks = refinedAttachedNeighborsTasks.data;
        refinedTasksSize = refinedAttachedNeighborsTasks.size;
        refinedResults = d_attachedNeighborsResults.data;
        break;
    case neighbour_type_enum::not_neighbors:
        refinedTasks = refinedNotNeighborsTasks.data;
        refinedTasksSize = refinedNotNeighborsTasks.size;
        refinedResults = d_notNeighborsResults.data;
        break;
    }

    if(results.size){
        int blocks = blocksForSize(refinedTasksSize);
        kSumIntegrationResults<<<blocks, gpuThreads>>>(refinedTasksSize, results.data, refinedResults, refinedTasks);
    }
}

__device__ double4 integrate4D(const double4 *functionValues)
{
    double4 res = { 0.0, 0.0, 0.0, 0.0 };
    for(int i = 0; i < c_GaussPointsNumber; ++i)
        res += c_GaussPointsWeights[i] * functionValues[i];
    
    return res;
}

__device__ void calculateQuadraturePoints(Point3 *quadraturePoints, const Point3 *vertices, const int3 &triangle)
{
    Point3 triangleVertices[3];
    triangleVertices[0] = vertices[triangle.x];
    triangleVertices[1] = vertices[triangle.y];
    triangleVertices[2] = vertices[triangle.z];

    for(int i = 0; i < c_GaussPointsNumber; ++i){
        Point3 res = { 0.0, 0.0, 0.0 };
        const Point3 Lcoordinates = c_GaussPointsCoordinates[i];

        for(int j = 0; j < 3; ++j)
            res += *(&Lcoordinates.x + j) * triangleVertices[j];

        quadraturePoints[i] = res;
    }
}
