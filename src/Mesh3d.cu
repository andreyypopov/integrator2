#include "hip/hip_runtime.h"
#include "Mesh3d.cuh"

#include "common/cuda_memory.cuh"
#include "common/constants.h"

#include <fstream>

__global__ void kCalculateCellNormal(int n, const Point3 *vertices, const int3 *cells, Point3 *normals){
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if(idx < n){
        const int3 triangle = cells[idx];
        const Point3 v12 = vertices[triangle.y] - vertices[triangle.x];
        const Point3 v13 = vertices[triangle.z] - vertices[triangle.x];

        normals[idx] = normalize(cross(v12, v13)); 
    }
}

__global__ void kCalculateCellCenter(int n, const Point3 *vertices, const int3 *cells, Point3 *centers){
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if(idx < n){
        const int3 triangle = cells[idx];
        centers[idx] = CONSTANTS::ONE_THIRD * (vertices[triangle.x] + vertices[triangle.y] + vertices[triangle.z]);
    }
}

__global__ void kCalculateCellMeasure(int n, const Point3 *vertices, const int3 *cells, double *measures){
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if(idx < n){
        const int3 triangle = cells[idx];
        const Point3 v12 = vertices[triangle.y] - vertices[triangle.x];
        const Point3 v13 = vertices[triangle.z] - vertices[triangle.x];

        measures[idx] = vector_length(cross(v12, v13)) * 0.5; 
    }
}

__global__ void kDetermineNeighborType(int n, const int3 *cells, int2 *simpleNeighbors, int *simpleNeighborsNum,
    int2 *attachedNeighbors, int *attachedNeighborsNum, int2 *notNeighbors, int *notNeighborsNum){
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int tri1idx = idx / n;
    int tri2idx = idx % n;

    if(tri1idx < n && tri2idx < n && tri1idx < tri2idx){
        unsigned int commonPoints = 0;
        int3 tri1 = cells[tri1idx];
        int3 tri2 = cells[tri2idx];

        if(tri1.x == tri2.x || tri1.x == tri2.y || tri1.x == tri2.z)
            ++commonPoints;

        if(tri1.y == tri2.x || tri1.y == tri2.y || tri1.y == tri2.z)
            ++commonPoints;

        if(tri1.z == tri2.x || tri1.z == tri2.y || tri1.z == tri2.z)
            ++commonPoints;
        
        if(commonPoints == 0){
            int pos = atomicAdd(notNeighborsNum, 1);
            notNeighbors[pos] = int2({ tri1idx, tri2idx });
        }
        
        if(commonPoints == 1){
            int pos = atomicAdd(simpleNeighborsNum, 1);
            simpleNeighbors[pos] = int2({ tri1idx, tri2idx });
        }

        if(commonPoints == 2){
            int pos = atomicAdd(attachedNeighborsNum, 1);
            attachedNeighbors[pos] = int2({ tri1idx, tri2idx });
        }
    }
}

Mesh3D::~Mesh3D(){
    free_device(d_simpleNeighborsNum);
    free_device(d_attachedNeighborsNum);
    free_device(d_notNeighborsNum);
}

bool Mesh3D::loadMeshFromFile(const std::string &filename, double scale)
{
    std::ifstream meshFile(filename);

    if(meshFile.is_open()){
        int numVertices, numCells;
        int tmp;

        meshFile >> numVertices >> numCells;

        std::vector<Point3> hostVertices;
        std::vector<int3> hostCells;

        hostVertices.reserve(numVertices);
        hostCells.reserve(numCells);

        for(int i = 0; i < numVertices; ++i){
            Point3 vertex;
            meshFile >> tmp >> vertex.x >> vertex.y >> vertex.z;
            hostVertices.push_back(scale * vertex);
        }

        while(!meshFile.eof()){
            meshFile >> tmp >> tmp;
            if(tmp == 203){ //encountered a triangle
                int3 triangle;
                meshFile >> triangle.x >> triangle.y >> triangle.z;
                
                //indices of vertices are base-1 in the imported files
                triangle.x -= 1;
                triangle.y -= 1;
                triangle.z -= 1;

                hostCells.push_back(triangle);
            } else {        //encountered an entity of another type
                numCells -= 1;
                meshFile >> tmp >> tmp;
            }
        }

        meshFile.close();

        vertices.allocate(numVertices);
        cells.allocate(numCells);
        cellNormals.allocate(numCells);
        cellCenters.allocate(numCells);
        cellMeasures.allocate(numCells);

        copy_h2d(hostVertices.data(), vertices.data, vertices.size);
        copy_h2d(hostCells.data(), cells.data, cells.size);

        printf("Loaded mesh with %d vertices and %d cells\n", numVertices, numCells);

        return true;
    } else {
        printf("Error while opening the file\n");
        return false;
    }
}

void Mesh3D::prepareMesh(){
    calculateNormals();
    calculateCenters();
    calculateMeasures();
    fillNeightborsLists();

    hipDeviceSynchronize();
}

void Mesh3D::calculateNormals(){
    unsigned int blocks = blocksForSize(cells.size);
    kCalculateCellNormal<<<blocks, gpuThreads>>>(cells.size, vertices.data, cells.data, cellNormals.data);
}

void Mesh3D::calculateCenters(){
    unsigned int blocks = blocksForSize(cells.size);
    kCalculateCellCenter<<<blocks, gpuThreads>>>(cells.size, vertices.data, cells.data, cellCenters.data);
}

void Mesh3D::calculateMeasures(){
    unsigned int blocks = blocksForSize(cells.size);
    kCalculateCellMeasure<<<blocks, gpuThreads>>>(cells.size, vertices.data, cells.data, cellMeasures.data);
}

void Mesh3D::fillNeightborsLists(){
    allocate_device(&d_simpleNeighborsNum, 1);
    allocate_device(&d_attachedNeighborsNum, 1);
    allocate_device(&d_notNeighborsNum, 1);
    zero_value_device(d_simpleNeighborsNum, 1);
    zero_value_device(d_attachedNeighborsNum, 1);
    zero_value_device(d_notNeighborsNum, 1);

    simpleNeighbors.allocate(cells.size * CONSTANTS::MAX_SIMPLE_NEIGHBORS_PER_VERTEX / 2);    // number of triangles * (3 neighbors * 3 vertices per triangle) / 2 (discard duplicated pairs)
    attachedNeighbors.allocate(cells.size * 3 / 2);  // number of triangles * (3 vertices edges per triangle) / 2 (discard duplicated pairs)
    notNeighbors.allocate(cells.size * cells.size / 2);  // all pairs of triangls

    unsigned int blocks = blocksForSize(cells.size * cells.size);
    kDetermineNeighborType<<<blocks, gpuThreads>>>(cells.size, cells.data, simpleNeighbors.data, d_simpleNeighborsNum, attachedNeighbors.data, d_attachedNeighborsNum, notNeighbors.data, d_notNeighborsNum);
    copy_d2h(d_simpleNeighborsNum, &simpleNeighbors.size, 1);
    copy_d2h(d_attachedNeighborsNum, &attachedNeighbors.size, 1);
    copy_d2h(d_notNeighborsNum, &notNeighbors.size, 1);

    printf("Found %d pairs of simple neighbors and %d pairs of attached neighbors, %d pairs are not neighbors\n", simpleNeighbors.size, attachedNeighbors.size, notNeighbors.size);
}

void exportMeshToObj(const std::string &filename, const std::vector<Point3> &vertices, const std::vector<int3> &cells)
{
    std::ofstream outputFile(filename.c_str());

    for(const auto &pt : vertices)
        outputFile << "v " << pt.x << " " << pt.y << " " << pt.z << std::endl;

    for(const auto &triangle : cells)
        outputFile << "f " << triangle.x + 1 << " " << triangle.y + 1 << " " << triangle.z + 1 << std::endl;

    outputFile.close();
}
