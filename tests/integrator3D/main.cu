#include "hip/hip_runtime.h"
#include <iostream>

#include "../../src/Mesh3d.cuh"
#include "../../src/NumericalIntegrator3d.cuh"
#include "../../src/evaluators/evaluatorJ3DK.cuh"

#include "../../thirdparty/optionparser-1.7/src/optionparser.h"

struct Arg: public option::Arg
{
  static void printError(const char* msg1, const option::Option& opt, const char* msg2)
  {
    fprintf(stderr, "%s", msg1);
    fwrite(opt.name, opt.namelen, 1, stderr);
    fprintf(stderr, "%s", msg2);
  }

  static option::ArgStatus Unknown(const option::Option& option, bool msg)
  {
    if (msg) printError("Unknown option '", option, "'\n");
    return option::ARG_ILLEGAL;
  }

  static option::ArgStatus Required(const option::Option& option, bool msg)
  {
    if (option.arg != 0)
      return option::ARG_OK;

    if (msg) printError("Option '", option, "' requires an argument\n");
    return option::ARG_ILLEGAL;
  }

  static option::ArgStatus NonEmpty(const option::Option& option, bool msg)
  {
    if (option.arg != 0 && option.arg[0] != 0)
      return option::ARG_OK;

    if (msg) printError("Option '", option, "' requires a non-empty argument\n");
    return option::ARG_ILLEGAL;
  }

  static option::ArgStatus Numeric(const option::Option& option, bool msg)
  {
    char* endptr = 0;
    if (option.arg != 0 && strtod(option.arg, &endptr)){};
    if (endptr != option.arg && *endptr == 0)
      return option::ARG_OK;

    if (msg) printError("Option '", option, "' requires a numeric argument\n");
    return option::ARG_ILLEGAL;
  }
};

enum optionIndex { UNKNOWN, HELP, MESHFILENAME, SCALE, EXPORTMESH, EXPORTRESULTS };

const option::Descriptor usage[] = 
{
    { UNKNOWN,          0, "",  "",             Arg::Unknown,   "USAGE: integrator2test3D [options]\n\n"
                                                                "Options:"},
    { HELP,             0, "h", "help",         Arg::None,      "   -h, \t--help  \tPrint usage and exit." },
    { MESHFILENAME,     0, "f", "meshfile",     Arg::NonEmpty,  "   -f <arg>, \t--meshfile=<arg> \tInput mesh file name." },
    { SCALE,            0, "s", "scale",        Arg::Numeric,   "   -s <arg>, \t--scale=<arg> \tMesh scale factor." },
    { EXPORTMESH,       0, "",  "exportmesh",   Arg::None,      "   \t--exportmesh \tExport original and refined meshes to OBJ files."},
    { EXPORTRESULTS,    0, "",  "exportresults",Arg::None,      "   \t--exportresults \tExport results of integration to text files."},
    { 0, 0, 0, 0, 0, 0 }
};

int main(int argc, char *argv[]){
    argc -= 1;
    argv += 1;

    option::Stats   stats(usage, argc, argv);
    option::Option* options = new option::Option[stats.options_max];
    option::Option* buffer = new option::Option[stats.buffer_max];
    option::Parser  parse(usage, argc, argv, options, buffer);

    if(parse.error())
        return EXIT_FAILURE;

    if(options[HELP] || argc == 0){
        option::printUsage(std::cout, usage);
        return EXIT_SUCCESS;
    }
    
    if(!options[MESHFILENAME]){
        printf("No input file with mesh specified. Exiting\n");
        return EXIT_FAILURE;
    }
    
    const std::string meshfilename(options[MESHFILENAME].arg);
    double scale = 1.0;
    if(options[SCALE])
        scale = std::stod(options[SCALE].arg);
    
    Mesh3D mesh;
    if(!mesh.loadMeshFromFile(meshfilename, scale))
        return EXIT_FAILURE;

    mesh.prepareMesh();

    std::vector<Point3> vertices;
    std::vector<int3> cells;

    vertices.resize(mesh.getVertices().size);
    copy_d2h(mesh.getVertices().data, vertices.data(), mesh.getVertices().size);
    cells.resize(mesh.getCells().size);
    copy_d2h(mesh.getCells().data, cells.data(), mesh.getCells().size);

    if(options[EXPORTMESH])
        exportMeshToObj("OriginalMesh.obj", vertices, cells);

    NumericalIntegrator3D numIntegrator(mesh, qf3D13);
    EvaluatorJ3DK evaluator(mesh, numIntegrator);
    evaluator.setFixedRefinementLevel(3);

    evaluator.runAllPairs();

    if(options[EXPORTRESULTS]){
        evaluator.outputResultsToFile(neighbour_type_enum::simple_neighbors);
        evaluator.outputResultsToFile(neighbour_type_enum::attached_neighbors);
        evaluator.outputResultsToFile(neighbour_type_enum::not_neighbors);
    }

    vertices.resize(numIntegrator.getRefinedVertices().size);
    copy_d2h(numIntegrator.getRefinedVertices().data, vertices.data(), numIntegrator.getRefinedVertices().size);
    cells.resize(numIntegrator.getRefinedCells().size);
    copy_d2h(numIntegrator.getRefinedCells().data, cells.data(), numIntegrator.getRefinedCells().size);

    if(options[EXPORTMESH])
        exportMeshToObj("RefinedMesh.obj", vertices, cells);

    return EXIT_SUCCESS;
}